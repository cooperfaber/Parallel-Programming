#include "hip/hip_runtime.h"


__global__ void blur(double * d_input, double * d_output, int d_size) {
    int start = threadIdx.x;
    //cover first index
    if(start == 0)start = 1;
    //blur
    for (int i = start; i < d_size-1; i+= blockDim.x) {
        d_output[i] = (d_input[i-1] + d_input[i] + d_input[i+1]) / 3;
    }
}



int main() {
 double *input = new double[SIZE];
 double *output = new double[SIZE];
 
 // input initialization (not shown)

 //copy inputs to graphics card
 hipMalloc(&d_input,SIZE*sizeof(double));
 hipMalloc(&d_output,SIZE*sizeof(double));
 hipMalloc(&d_size,sizeof(int));

 hipMemcpy(d_input, input, SIZE*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(d_output, output, SIZE*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(d_size, SIZE, sizeof(int),hipMemcpyHostToDevice);

 vector_add<<<1,512>>>blur(d_input, d_output, d_size);
 // check results in output (not shown)
 return 0;
}